#include "hip/hip_runtime.h"
//
// Created by lwilkinson on 11/4/21.
//

#include <math.h>
#include <iostream>
#include "hip/hip_runtime.h"
#include "hipblas.h"
#include "cupti_profiler.h"
#include <stdlib.h>
#include <iostream>
#include <vector>
#include <algorithm>
#include <array>
#include <numeric>
#include "common/utils/matrix_utils.h"
#include "sputnik/sputnik.h"

// cuBLAS API errors
static const char *_cudaGetErrorEnum(hipblasStatus_t error)
{
  switch (error)
  {
    case HIPBLAS_STATUS_SUCCESS:
      return "HIPBLAS_STATUS_SUCCESS";

    case HIPBLAS_STATUS_NOT_INITIALIZED:
      return "HIPBLAS_STATUS_NOT_INITIALIZED";

    case HIPBLAS_STATUS_ALLOC_FAILED:
      return "HIPBLAS_STATUS_ALLOC_FAILED";

    case HIPBLAS_STATUS_INVALID_VALUE:
      return "HIPBLAS_STATUS_INVALID_VALUE";

    case HIPBLAS_STATUS_ARCH_MISMATCH:
      return "HIPBLAS_STATUS_ARCH_MISMATCH";

    case HIPBLAS_STATUS_MAPPING_ERROR:
      return "HIPBLAS_STATUS_MAPPING_ERROR";

    case HIPBLAS_STATUS_EXECUTION_FAILED:
      return "HIPBLAS_STATUS_EXECUTION_FAILED";

    case HIPBLAS_STATUS_INTERNAL_ERROR:
      return "HIPBLAS_STATUS_INTERNAL_ERROR";
  }

  return "<unknown>";
}

#define CHECK_CUDA(func)                                                        \
    {                                                                           \
        hipError_t status = (func);                                            \
        if (status != hipSuccess)                                              \
        {                                                                       \
            printf("CUDA API failed at %s:%d with error: %s (%d)\n",            \
                   __FILE__, __LINE__, hipGetErrorString(status), status);     \
            return EXIT_FAILURE;                                                \
        }                                                                       \
    }

#define CHECK_CUSPARSE(func)                                                    \
    {                                                                           \
        hipsparseStatus_t status = (func);                                       \
        if (status != HIPSPARSE_STATUS_SUCCESS)                                  \
        {                                                                       \
            printf("CUSPARSE API failed at %s:%d with error: %s (%d)\n",        \
                   __FILE__, __LINE__, hipsparseGetErrorString(status), status); \
            return EXIT_FAILURE;                                                \
        }                                                                       \
    }

#define CHECK_CUBLAS(func)                                                    \
    {                                                                           \
        hipblasStatus_t status = (func);                                         \
        if (status != HIPBLAS_STATUS_SUCCESS)                                    \
        {                                                                       \
            printf("CUSPARSE API failed at %s:%d with error: %s (%d)\n",        \
                   __FILE__, __LINE__, _cudaGetErrorEnum(status), status);       \
            return EXIT_FAILURE;                                                \
        }                                                                       \
    }


using namespace std;

typedef struct Dense {
    int rows;
    int cols;
    float * values;

    Dense(int m, int n, float fill_val): rows(m), cols(n) {
      values = new float[rows * cols];
      for (int i = 0; i < rows * cols; i++) values[i] = fill_val;
    }

    Dense(int m, int n, float * values): rows(m), cols(n), values(values) {}

    float coeff(int i, int j) const {
      return values[i * cols + j];
    }

    inline void set_coeff(int i, int j, float val) {
      values[i * cols + j] = val;
    }

    void fill(float fill_val) {
      for (int i = 0; i < rows * cols; i++) values[i] = fill_val;
    }

} Dense;

struct Chunks {
  int * col_pattern;
  int * row_pattern;
  int * row_offsets;
};


// Taken from: https://stackoverflow.com/a/18856054
//__global__ void _codelet_multiply(const std::vector<> Dense A, Dense B, Dense C)
//{
//
//}

static const int TILE_DIM = 32;
static const int ITERATIONS = 10;

int cublas_multiply(hipStream_t& stream, hipEvent_t& start, hipEvent_t& stop, const Dense& A, const Dense& B, Dense& C)
{
    float alpha = 1.0, beta = 1.0;
    hipblasHandle_t handle;
    CHECK_CUBLAS(hipblasCreate(&handle));
    hipEventRecord(start, stream);

    // https://stackoverflow.com/a/56064726
    CHECK_CUBLAS(hipblasSgemm(handle,
      HIPBLAS_OP_N, HIPBLAS_OP_N,
      C.cols, C.rows, A.cols, /* m, n, k */
      &alpha,
      B.values, B.cols, /* *A, lda */
      A.values, A.cols, /* *B, lda */
      &beta,
      C.values, C.cols))
    hipEventRecord(stop, stream);
    return 0;
}

// Taken from: https://stackoverflow.com/a/18856054
__global__ void _dense_multiply(const Dense& A, const Dense& B, const Dense& C)
{
  float CValue = 0;

  int Row = blockIdx.y*TILE_DIM + threadIdx.y;
  int Col = blockIdx.x*TILE_DIM + threadIdx.x;

  __shared__ float As[TILE_DIM][TILE_DIM];
  __shared__ float Bs[TILE_DIM][TILE_DIM];

  for (int k = 0; k < (A.cols + TILE_DIM - 1)/TILE_DIM; k++) {

    if (k*TILE_DIM + threadIdx.x < A.cols && Row < A.rows)
      As[threadIdx.y][threadIdx.x] = A.values[Row * A.cols + k*TILE_DIM + threadIdx.x];
    else
      As[threadIdx.y][threadIdx.x] = 0.0;

    if (k*TILE_DIM + threadIdx.y < B.rows && Col < B.cols)
      Bs[threadIdx.y][threadIdx.x] = B.values[(k*TILE_DIM + threadIdx.y) * B.cols + Col];
    else
      Bs[threadIdx.y][threadIdx.x] = 0.0;

    __syncthreads();

    for (int n = 0; n < TILE_DIM; ++n)
      CValue += As[threadIdx.y][n] * Bs[n][threadIdx.x];

    __syncthreads();
  }


  if (Row < C.rows && Col < C.cols) {
    C.values[(Row * C.cols) + Col] = CValue;
  }
}

int dense_multiply(hipStream_t& stream, hipEvent_t& start, hipEvent_t& stop, const Dense& A, const Dense& B, const Dense& C) {
  dim3 block_dim(TILE_DIM, TILE_DIM);
  dim3 grid_dim((C.cols + (TILE_DIM - 1)) / TILE_DIM, (C.rows + (TILE_DIM - 1)) / TILE_DIM);

  hipEventRecord(start, stream);
  _dense_multiply<<<grid_dim, block_dim, 0, stream>>>(A, B, C);
  CHECK_CUDA(hipGetLastError());
  hipEventRecord(stop, stream);

  return 0;
}

static const int BLOCK_MULTIPLY_TILE_K = 32;
static const int BLOCK_MULTIPLY_MAX_COLS_PER_BLOCK = 16;
static const int BLOCK_MULTIPLY_MAX_ROWS_PER_BLOCK = 8;

typedef struct {
  int * col_pattern;
  int col_pattern_len;
  int * rows;
  int num_rows;
  float * row_segment_values;

  float coeff(int row, int col_pattern_idx) const {
    return row_segment_values[row * col_pattern_len + col_pattern_idx];
  }
} Block;

__device__ __forceinline__ float coeff(const Dense& A, int i, int j) {
  return A.values[i * A.cols + j];
}

__device__ __forceinline__ float atomic_add_coeff(Dense& A, int i, int j, float val) {
  atomicAdd(&A.values[i * A.cols + j], val);
}

__device__ __forceinline__ float coeff(const Block& A, int row, int col_pattern_idx) {
  return A.row_segment_values[row * A.col_pattern_len + col_pattern_idx];
}

// Taken from: https://stackoverflow.com/a/18856054
__global__ void _block_multiply(const Block * blocks, int num_blocks, const CSR<float> A, const Dense B, Dense C) {
  int block_idx = blockIdx.x;

  const Block *block = &blocks[block_idx];

  // TODO: make BLOCK_MULTIPLY_MAX_COLS_PER_BLOCK dynamic
  __shared__ float A_s[BLOCK_MULTIPLY_MAX_ROWS_PER_BLOCK * BLOCK_MULTIPLY_MAX_COLS_PER_BLOCK];
  __shared__ float B_s[BLOCK_MULTIPLY_MAX_COLS_PER_BLOCK][BLOCK_MULTIPLY_TILE_K];
  __shared__ float C_s[BLOCK_MULTIPLY_MAX_ROWS_PER_BLOCK][BLOCK_MULTIPLY_TILE_K];

  int non_zeros = block->num_rows * block->col_pattern_len;
  int full_loads = non_zeros / BLOCK_MULTIPLY_TILE_K;
  int full_loads_end = full_loads * BLOCK_MULTIPLY_TILE_K;
  int partial_load = non_zeros - full_loads_end;

  for (int i = 0; i < full_loads; i ++) {
    A_s[i * BLOCK_MULTIPLY_TILE_K + threadIdx.x] = block->row_segment_values[i * BLOCK_MULTIPLY_TILE_K + threadIdx.x];
  }

  if (threadIdx.x < partial_load) {
    A_s[full_loads_end + threadIdx.x] = block->row_segment_values[full_loads_end + threadIdx.x];
  }

  for (int k = 0; k < B.cols; k+= BLOCK_MULTIPLY_TILE_K) {
    __syncthreads();

    for (int row_idx = 0; row_idx < block->num_rows; row_idx++) {
      C_s[row_idx][threadIdx.x] = 0;
    }

    for (int col_pattern_idx = 0; col_pattern_idx < block->col_pattern_len; col_pattern_idx++) {
      int col = block->col_pattern[col_pattern_idx];

      // Cooperative loading
      static_assert(BLOCK_MULTIPLY_TILE_K == 32);
      B_s[col_pattern_idx][threadIdx.x] = coeff(B, col, k + threadIdx.x);

    }

    __syncthreads();

    for (int row_idx = 0; row_idx < block->num_rows; row_idx++) {
      for (int col_pattern_idx = 0; col_pattern_idx < block->col_pattern_len; col_pattern_idx++) {
        C_s[row_idx][threadIdx.x] += B_s[col_pattern_idx][threadIdx.x] * A_s[row_idx * block->col_pattern_len + col_pattern_idx];
      }
    }

    __syncthreads();

    for (int row_idx = 0; row_idx < block->num_rows; row_idx++) {
      atomic_add_coeff(C, block->rows[row_idx], k + threadIdx.x, C_s[row_idx][threadIdx.x]);
    }

    __syncthreads();
  }
}


int codelet_multiply(hipStream_t& stream, hipEvent_t& start, hipEvent_t& stop, const Block * blocks, size_t num_blocks,
                     const CSR<float>& A_h, const CSR<float>& A, const Dense& B, Dense& C) {
  dim3 block_dim(BLOCK_MULTIPLY_TILE_K, 1);
  dim3 grid_dim(num_blocks, 1);

  hipEventRecord(start, stream);
  _block_multiply<<<grid_dim, block_dim, 0, stream>>>(blocks, num_blocks, A, B, C);
  CHECK_CUDA(hipGetLastError());
  hipEventRecord(stop, stream);

  return 0;
}



int sgk_multiply(hipStream_t& stream, hipEvent_t& start, hipEvent_t& stop, const CSR<float>& A_h, const CSR<float>& A, const Dense& B, Dense& C) {
  float* bias = nullptr;

  // Sort rows - Copied from sputnik
  // Create our unsorted row indices.
  std::vector<int> swizzle_staging(A_h.rows);
  std::iota(swizzle_staging.begin(), swizzle_staging.end(), 0);

  // Argsort the row indices based on their length.
  std::sort(swizzle_staging.begin(), swizzle_staging.end(),
            [&A_h](int idx_a, int idx_b) {
              int length_a = A_h.row_offsets[idx_a + 1] - A_h.row_offsets[idx_a];
              int length_b = A_h.row_offsets[idx_b + 1] - A_h.row_offsets[idx_b];
              return length_a > length_b;
            });

  int *swizzle_d;
  CHECK_CUDA(hipMalloc((void **)&swizzle_d, sizeof(int) * A_h.rows));
  CHECK_CUDA(hipMemcpy(swizzle_d, swizzle_staging.data(), sizeof(int) * A_h.rows, hipMemcpyHostToDevice));

  hipDeviceSynchronize();

  hipEventRecord(start, stream);
  CHECK_CUDA(sputnik::CudaSpmmBiasRelu(A.rows, A.cols, B.cols, A.nnz, swizzle_d,
                                       A.values, A.row_offsets, A.col_indices,
                                       B.values,
                                       bias,
                                       C.values,
                                       stream))
  hipEventRecord(stop, stream);
  CHECK_CUDA(hipFree(swizzle_d))
  return 0;
}

int run_kernel(const Dense& A, const Dense& B, const Dense& C, const std::string& name,
                int(*kernel)(hipStream_t& stream, hipEvent_t& start, hipEvent_t& stop, const Dense& A, const Dense& B, Dense& C)) {
  float *A_values_d, *B_values_d, *C_values_d;

  CHECK_CUDA(hipMalloc(&A_values_d, A.rows * A.cols * sizeof(float)));
  CHECK_CUDA(hipMalloc(&B_values_d, B.rows * B.cols * sizeof(float)));
  CHECK_CUDA(hipMalloc(&C_values_d, C.rows * C.cols * sizeof(float)));

  CHECK_CUDA(hipMemcpy(A_values_d, A.values, A.rows * A.cols * sizeof(float), hipMemcpyHostToDevice));
  CHECK_CUDA(hipMemcpy(B_values_d, B.values, B.rows * B.cols * sizeof(float), hipMemcpyHostToDevice));

  Dense A_d = A;
  Dense B_d = B;
  Dense C_d = C;

  A_d.values = A_values_d;
  B_d.values = B_values_d;
  C_d.values = C_values_d;

  hipEvent_t start, stop;
  hipEventCreate(&start);
  hipEventCreate(&stop);

  hipStream_t stream = NULL;
  CHECK_CUDA(hipStreamCreate(&stream))

  hipDeviceSynchronize();

  float total_time = 0;
  for (int iter = 0; iter < ITERATIONS + 1; iter++) {
    CHECK_CUDA(hipMemset(C_values_d, 0, C.rows * C.cols * sizeof(float)));
    hipDeviceSynchronize();

    kernel(stream, start, stop, A_d, B_d, C_d);
    hipEventSynchronize(stop);
    CHECK_CUDA(hipGetLastError());

    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);

    if (iter >= 1) total_time += milliseconds; // Skip a warm up
  }

  std::cout << name << " took " << total_time / ITERATIONS << "ms (avg)" << std::endl;

  CHECK_CUDA(hipMemcpy(C.values, C_values_d, C.rows * C.cols * sizeof(float), hipMemcpyDeviceToHost));

  CHECK_CUDA(hipFree(A_values_d))
  CHECK_CUDA(hipFree(B_values_d))
  CHECK_CUDA(hipFree(C_values_d))

  return 0;
}

int run_kernel(const CSR<float>& A, const Dense& B, Dense& C, const std::string& name,
               int(*kernel)(hipStream_t& stream, hipEvent_t& start, hipEvent_t& stop, const CSR<float>& A_h, const CSR<float>& A, const Dense& B, Dense& C)) {
  float *A_values_d, *B_values_d, *C_values_d;
  int *A_row_offsets_d, *A_col_indices_d;

  CHECK_CUDA(hipMalloc(&A_values_d, A.nnz * sizeof(float)));
  CHECK_CUDA(hipMalloc(&A_row_offsets_d, (A.rows + 1) * sizeof(int)));
  CHECK_CUDA(hipMalloc(&A_col_indices_d, A.nnz * sizeof(int)));

  CHECK_CUDA(hipMalloc(&B_values_d, B.rows * B.cols * sizeof(float)));
  CHECK_CUDA(hipMalloc(&C_values_d, C.rows * C.cols * sizeof(float)));

  CHECK_CUDA(hipMemcpy(A_values_d, A.values, A.nnz * sizeof(float), hipMemcpyHostToDevice));
  CHECK_CUDA(hipMemcpy(A_row_offsets_d, A.row_offsets, (A.rows + 1) * sizeof(int), hipMemcpyHostToDevice));
  CHECK_CUDA(hipMemcpy(A_col_indices_d, A.col_indices, A.nnz * sizeof(int), hipMemcpyHostToDevice));

  CHECK_CUDA(hipMemcpy(B_values_d, B.values, B.rows * B.cols * sizeof(float), hipMemcpyHostToDevice));

  CSR<float> A_d = A;
  Dense B_d = B;
  Dense C_d = C;

  A_d.values = A_values_d;
  A_d.row_offsets = A_row_offsets_d;
  A_d.col_indices = A_col_indices_d;
  B_d.values = B_values_d;
  C_d.values = C_values_d;

  hipDeviceSynchronize();

  hipEvent_t start, stop;
  hipEventCreate(&start);
  hipEventCreate(&stop);

  hipStream_t stream = NULL;
  CHECK_CUDA(hipStreamCreate(&stream))

  hipDeviceSynchronize();

  float total_time = 0;
  for (int iter = 0; iter < ITERATIONS + 1; iter++) {
    CHECK_CUDA(hipMemset(C_values_d, 0, C.rows * C.cols * sizeof(float)));
    hipDeviceSynchronize();

    kernel(stream, start, stop, A, A_d, B_d, C_d);
    hipEventSynchronize(stop);
    CHECK_CUDA(hipGetLastError());

    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);

    if (iter >= 1) total_time += milliseconds; // Skip a warm up
  }

  std::cout << name << " took " << total_time / ITERATIONS << "ms (avg)" << std::endl;

  CHECK_CUDA(hipMemcpy(C.values, C_values_d, C.rows * C.cols * sizeof(float), hipMemcpyDeviceToHost));

  CHECK_CUDA(hipFree(A_values_d));
  CHECK_CUDA(hipFree(A_row_offsets_d));
  CHECK_CUDA(hipFree(A_col_indices_d));

  CHECK_CUDA(hipFree(B_values_d))
  CHECK_CUDA(hipFree(C_values_d))

  return 0;
}

int run_kernel(const std::vector<Block> &blocks, const CSR<float>& A, const Dense& B, Dense& C, const std::string& name,
               int(*kernel)(hipStream_t& stream, hipEvent_t& start, hipEvent_t& stop, const Block * blocks, size_t num_blocks, const CSR<float>& A_h, const CSR<float>& A, const Dense& B, Dense& C)) {
  float *A_values_d, *B_values_d, *C_values_d;
  int *A_row_offsets_d, *A_col_indices_d;
  std::vector<Block> blocks_d_temp(blocks);
  Block* blocks_d;

  CHECK_CUDA(hipMalloc(&A_values_d, A.nnz * sizeof(float)));
  CHECK_CUDA(hipMalloc(&A_row_offsets_d, (A.rows + 1) * sizeof(int)));
  CHECK_CUDA(hipMalloc(&A_col_indices_d, A.nnz * sizeof(int)));

  for (int i = 0; i < blocks.size(); i++) {
    CHECK_CUDA(hipMalloc(&blocks_d_temp[i].rows, blocks_d_temp[i].num_rows * sizeof(int)));
    CHECK_CUDA(hipMemcpy(blocks_d_temp[i].rows, blocks[i].rows, blocks_d_temp[i].num_rows * sizeof(int), hipMemcpyHostToDevice));

    CHECK_CUDA(hipMalloc(&blocks_d_temp[i].col_pattern, blocks_d_temp[i].col_pattern_len * sizeof(int)));
    CHECK_CUDA(hipMemcpy(blocks_d_temp[i].col_pattern, blocks[i].col_pattern, blocks_d_temp[i].col_pattern_len * sizeof(int), hipMemcpyHostToDevice));

    CHECK_CUDA(hipMalloc(&blocks_d_temp[i].row_segment_values, blocks_d_temp[i].num_rows * blocks_d_temp[i].col_pattern_len * sizeof(float)));
    CHECK_CUDA(hipMemcpy(blocks_d_temp[i].row_segment_values, blocks[i].row_segment_values, blocks_d_temp[i].num_rows * blocks_d_temp[i].col_pattern_len * sizeof(float), hipMemcpyHostToDevice));
  }

  CHECK_CUDA(hipMalloc(&blocks_d, blocks_d_temp.size() * sizeof(Block)));
  CHECK_CUDA(hipMemcpy(blocks_d, blocks_d_temp.data(), blocks_d_temp.size() * sizeof(Block), hipMemcpyHostToDevice));

  CHECK_CUDA(hipMalloc(&B_values_d, B.rows * B.cols * sizeof(float)));
  CHECK_CUDA(hipMalloc(&C_values_d, C.rows * C.cols * sizeof(float)));

  CHECK_CUDA(hipMemcpy(A_values_d, A.values, A.nnz * sizeof(float), hipMemcpyHostToDevice));
  CHECK_CUDA(hipMemcpy(A_row_offsets_d, A.row_offsets, (A.rows + 1) * sizeof(int), hipMemcpyHostToDevice));
  CHECK_CUDA(hipMemcpy(A_col_indices_d, A.col_indices, A.nnz * sizeof(int), hipMemcpyHostToDevice));

  CHECK_CUDA(hipMemcpy(B_values_d, B.values, B.rows * B.cols * sizeof(float), hipMemcpyHostToDevice));

  CSR<float> A_d = A;
  Dense B_d = B;
  Dense C_d = C;

  A_d.values = A_values_d;
  A_d.row_offsets = A_row_offsets_d;
  A_d.col_indices = A_col_indices_d;
  B_d.values = B_values_d;
  C_d.values = C_values_d;

  hipDeviceSynchronize();

  hipEvent_t start, stop;
  hipEventCreate(&start);
  hipEventCreate(&stop);

  hipStream_t stream = NULL;
  CHECK_CUDA(hipStreamCreate(&stream))

  hipDeviceSynchronize();

  float total_time = 0;
  for (int iter = 0; iter < ITERATIONS + 1; iter++) {
    CHECK_CUDA(hipMemset(C_values_d, 0, C.rows * C.cols * sizeof(float)));
    hipDeviceSynchronize();

    kernel(stream, start, stop, blocks_d, blocks.size(), A, A_d, B_d, C_d);
    hipEventSynchronize(stop);
    CHECK_CUDA(hipGetLastError());

    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);

    if (iter >= 1) total_time += milliseconds; // Skip a warm up
  }

  std::cout << name << " took " << total_time / ITERATIONS << "ms (avg)" << std::endl;

  CHECK_CUDA(hipMemcpy(C.values, C_values_d, C.rows * C.cols * sizeof(float), hipMemcpyDeviceToHost));

  CHECK_CUDA(hipFree(A_values_d));
  CHECK_CUDA(hipFree(A_row_offsets_d));
  CHECK_CUDA(hipFree(A_col_indices_d));

  for (int i = 0; i < blocks.size(); i++) {
    CHECK_CUDA(hipFree(blocks_d_temp[i].rows));
    CHECK_CUDA(hipFree(blocks_d_temp[i].col_pattern));
    CHECK_CUDA(hipFree(blocks_d_temp[i].row_segment_values));
  }

  CHECK_CUDA(hipFree(B_values_d));
  CHECK_CUDA(hipFree(C_values_d));

  return 0;
}


typedef struct codelet {
  std::vector<int> row_offsets;
  std::vector<int> col_offsets;
} codelet_t;

static const inline int partition_evenly(int size, int target) {
  int blocks = (size + target - 1) / target;
  return (size + blocks - 1) / blocks;
}

const std::vector<Block> gen_blocks(const std::vector<codelet_t>& codelets) {
  std::vector<Block> blocks;

  for (auto& codelet : codelets) {
    int cols_per_block = partition_evenly(codelet.col_offsets.size(), BLOCK_MULTIPLY_MAX_COLS_PER_BLOCK);
    int rows_per_block = partition_evenly(codelet.row_offsets.size(), BLOCK_MULTIPLY_MAX_ROWS_PER_BLOCK);

    for (int i = 0; i < codelet.row_offsets.size(); i+= rows_per_block) {
      for (int j = 0; j < codelet.col_offsets.size(); j+= cols_per_block) {
        int rows_in_block = std::min(codelet.row_offsets.size() - i, (size_t) rows_per_block);
        int cols_in_block = std::min(codelet.col_offsets.size() - i, (size_t) cols_per_block);

        Block blk;
        blk.num_rows = rows_in_block;
        blk.col_pattern_len = cols_in_block;
        blk.rows = new int[rows_in_block];
        blk.col_pattern = new int[cols_in_block];
        blk.row_segment_values = new float[rows_in_block * cols_in_block];

        for (int ii = 0; ii < rows_in_block; ii++) { blk.rows[ii] = codelet.row_offsets[ii + i]; }
        for (int jj = 0; jj < cols_in_block; jj++) { blk.col_pattern[jj] = codelet.col_offsets[jj + j]; }

        for (int i = 0; i < rows_in_block * cols_in_block; i++) {
          blk.row_segment_values[i] = 1;
        }

        blocks.push_back(blk);
      }
    }
  }

  return blocks;
}

CSR<float> gen_csr(size_t m, size_t n, const std::vector<codelet_t>& codelets) {
  std::vector<std::vector<int>> rows(m);

  for (auto& codelet : codelets) {
    for (auto& row_offset : codelet.row_offsets) {
      auto& row = rows[row_offset];
      if (codelet.col_offsets.size() > 0) {
        auto end_of_existing = row.size();

        row.insert(row.end(), codelet.col_offsets.begin(), codelet.col_offsets.end());
        std::inplace_merge(row.begin(), row.begin() + end_of_existing, row.end());
      }
    }
  }

  for (auto& row : rows) {
    auto last = std::unique(row.begin(), row.end());
    row.erase(last, row.end());
  }

  int nnz = 0; for (auto& row : rows) nnz += row.size();

  CSR<float> csr(m, n, nnz);
  int curr_offset = 0;
  csr.row_offsets[0] = 0;
  int i = 0;
  for (auto& row : rows) {
    std::copy( row.begin(), row.end(), &csr.col_indices[curr_offset]);
    curr_offset += row.size();
    csr.row_offsets[++i] = curr_offset;
  }

  for (int i = 0; i < nnz; i++) { csr.values[i] = 1; }

  return csr;
}

Dense csr_to_dense(const CSR<float> &csr) {
  float * out = new float[csr.rows * csr.cols];

  for (int i = 0; i < csr.rows; i ++) {
    for (int p = csr.row_offsets[i]; p < csr.row_offsets[i+1]; p++) {
      out[i * csr.cols + csr.col_indices[p]] = csr.values[p];
    }
  }

  return Dense(csr.rows, csr.cols, out);
}

void print_csr(const CSR<float> &csr) {
  for (int i = 0; i < csr.rows; i ++) {
    std::cout << i << ": ";
    for (int p = csr.row_offsets[i]; p < csr.row_offsets[i+1]; p++) {
      std::cout << csr.col_indices[p] << " ";
    }
    std::cout << std::endl;
  }
}

void print_dense(const Dense &dense) {
  for (int i = 0; i < dense.rows; i++) {
    for (int j = 0; j < dense.cols; j++) {
      std::cout << " " << dense.coeff(i, j);
    }
    std::cout << std::endl;
  }
}

void compare_dense(const Dense &A, const Dense &B) {
  assert(A.rows == A.rows && B.cols == B.cols);
  int total_errors = 0;
  for (int i = 0; i < A.rows; i++) {
    for (int j = 0; j < A.cols; j++) {
      if (A.coeff(i, j) != B.coeff(i, j)) {
        total_errors++;
        if (total_errors < 10) {
          printf("[ERROR] Mismatch at (%d, %d)\n", i, j);
        }
      }
    }
  }
  printf("[ERROR] Total Mismatch %d\n", total_errors);
}

std::vector<codelet_t> gen_checkerboard(int m, int n, int stride) {

  std::vector<codelet_t> checkerboard_codelets(2);

  checkerboard_codelets[0].col_offsets.reserve((n + 2) / stride);
  checkerboard_codelets[0].row_offsets.reserve((m + 2) / stride);
  checkerboard_codelets[1].col_offsets.reserve((n + 2) / stride);
  checkerboard_codelets[1].row_offsets.reserve((m + 2) / stride);

  for (int i = 0; i < n; i += stride) { checkerboard_codelets[0].col_offsets.push_back(i); }
  for (int i = 0; i < m; i += stride) { checkerboard_codelets[0].row_offsets.push_back(i); }

  for (int i = 1; i < n; i += stride) { checkerboard_codelets[1].col_offsets.push_back(i); }
  for (int i = 1; i < m; i += stride) { checkerboard_codelets[1].row_offsets.push_back(i); }

  return std::move(checkerboard_codelets);
}

int main() {
  const int batch_size = 256;
  const int A_rows = 1024;
  const int A_cols = 2048;

  std::cout << "Generating pattern ..." << std::endl;
  auto codelets = gen_checkerboard(A_rows, A_cols, 4);
  std::cout << "Constructing CSR matrix ..." << std::endl;
  CSR<float> csr = gen_csr(A_rows, A_cols, codelets);
  std::cout << "Sparsity " << (1.f - csr.nnz / float(csr.rows * csr.cols)) * 100 << "%" << std::endl;
  std::cout << "Generating dense version ..." << std::endl;
  auto A = csr_to_dense(csr);

  auto blocks = gen_blocks(codelets);

  Dense B(csr.cols, batch_size, 5.f);
  Dense C(csr.rows, batch_size, 0.f);
  Dense C_golden(csr.rows, batch_size, 0.f);

  std::cout << "Running kernels ..." << std::endl;


  //
  //  Run Kernels
  //


  run_kernel(A, B, C_golden, "cublas", cublas_multiply);

  run_kernel(csr, B, C, "sgk", sgk_multiply);
  compare_dense(C_golden, C);
  C.fill(0);
  run_kernel(blocks, csr, B, C, "codelets", codelet_multiply);
  compare_dense(C_golden, C);

  delete A.values;
}


//  for (int i = 0; i < csr.rows; i++) {
//    for (int p = csr.row_offsets[i]; p < csr.row_offsets[i+1]; p++) {
//      for (int j = 0; j < B.cols; j++) {
//        C_golden.values[i * C_golden.cols + j] += csr.values[p] * B.coeff(csr.col_indices[p], j);
//      }
//    }
//  }
