#include "hip/hip_runtime.h"
//
// Created by lwilkinson on 11/25/21.
//

#include "common/utils/cuda_utils.h"
#include "codlet_multiply.cuh"

using namespace CodeletMultiply;

__device__ __forceinline__
float coeff(const Dense& A, int i, int j) {
  return A.values[i * A.cols + j];
}

__device__ __forceinline__
float* coeff_ptr(const Dense& A, int i, int j) {
  return &A.values[i * A.cols + j];
}

__device__ __forceinline__
void vec_atomic_add_coeff(Dense& A, int i, int j, float4 val) {
  atomicAdd(&A.values[i * A.cols + j + 0], val.x);
  atomicAdd(&A.values[i * A.cols + j + 1], val.y);
  atomicAdd(&A.values[i * A.cols + j + 2], val.z);
  atomicAdd(&A.values[i * A.cols + j + 3], val.w);

//    *reinterpret_cast<float4*>(&A.values[i * A.cols + j + 0]) = val;

//    A.values[i * A.cols + j + 0] = val.x;
//    A.values[i * A.cols + j + 1] = val.y;
//    A.values[i * A.cols + j + 2] = val.z;
//    A.values[i * A.cols + j + 3] = val.w;
}

__device__ __forceinline__
float coeff(const Block& A, int row, int col_pattern_idx) {
  return A.row_segment_values[row * A.col_pattern_len + col_pattern_idx];
}

__device__ __forceinline__
void FMAA(float4 &accumulate, float a, float *__restrict__ b){
  auto b4 = *reinterpret_cast<float4 *>(b);

  accumulate.x += (a) * b4.x;
  accumulate.y += (a) * b4.y;
  accumulate.z += (a) * b4.z;
  accumulate.w += (a) * b4.w;
}

__device__ __forceinline__
void FMAA(float4 &accumulate, float a, float4 b){
  accumulate.x += (a) * b.x;
  accumulate.y += (a) * b.y;
  accumulate.z += (a) * b.z;
  accumulate.w += (a) * b.w;
}

template<int vector_size>
__device__ __forceinline__
void vector_load(float *__restrict__ dst, float *__restrict__ src);

template<>
__device__ __forceinline__
void vector_load<4>(float *__restrict__ dst, float *__restrict__ src) {
  *reinterpret_cast<float4 *>(dst) = *reinterpret_cast<float4 *>(src);
}

__device__ __forceinline__
float4  vector_load(float *__restrict__ src) {
  return *reinterpret_cast<float4 *>(src);
}

dim3 codelet_block_dim(BLOCK_X_DIM, BLOCK_Y_DIM);

// Taken from: https://stackoverflow.com/a/18856054
__global__ void _block_multiply(const Block * blocks, int num_blocks, const CSR<float> A, const Dense B, Dense C) {
  int block_idx = blockIdx.x;

  const Block *block = &blocks[block_idx];

  // TODO: make CODELET_MULTIPLY_MAX_COLS_PER_BLOCK dynamic
  __shared__ __align__(32) float A_s[MAX_ROWS_PER_BLOCK * MAX_COLS_PER_BLOCK];
  __shared__ __align__(32) float B_s[MAX_COLS_PER_BLOCK][TILE_K];

  int non_zeros = block->num_rows * block->col_pattern_len;
  int thd_idx_linear = threadIdx.x + threadIdx.y * blockDim.x;
  int thd_x_vec_offset = threadIdx.x * VECTOR_WIDTH;
  int thd_linear_vec_offset = thd_idx_linear * VECTOR_WIDTH;
  int block_size = blockDim.x * blockDim.y;


  int vector_load_block_width = block_size * VECTOR_WIDTH;
  int a_vector_loads_full_block = non_zeros / vector_load_block_width;
  int a_vector_loads_partial_block_start = a_vector_loads_full_block * vector_load_block_width;


  int a_vector_partial_load = non_zeros - a_vector_loads_partial_block_start;

  for (int i = 0; i < a_vector_loads_full_block; i ++) {

    int dst_idx = (i * block_size * VECTOR_WIDTH) + (thd_linear_vec_offset);
    int src_idx = (i * block_size * VECTOR_WIDTH) + (thd_linear_vec_offset);

    vector_load<4>(&A_s[src_idx], &block->row_segment_values[dst_idx]);
  }

  if (thd_idx_linear < a_vector_partial_load) {

    int dst_idx = (a_vector_loads_partial_block_start) + (thd_linear_vec_offset);
    int src_idx = (a_vector_loads_partial_block_start) + (thd_linear_vec_offset);

    vector_load<4>(&A_s[src_idx], &block->row_segment_values[dst_idx]);
  }

  __syncthreads();

  for (int k = 0; k < B.cols; k+= TILE_K) {

    for (int col_pattern_idx = threadIdx.y; col_pattern_idx < block->col_pattern_len; col_pattern_idx += blockDim.y) {
      int col = block->col_pattern[col_pattern_idx];

      vector_load<4>(&B_s[col_pattern_idx][thd_x_vec_offset], coeff_ptr(B, col, k + thd_x_vec_offset));
    }

    __syncthreads();

    static_assert(MAX_ROWS_PER_BLOCK / BLOCK_Y_DIM  == 2);
    float4 c0 = { .x = 0, .y = 0, .z = 0, .w = 0 };
    float4 c1 = { .x = 0, .y = 0, .z = 0, .w = 0 };

    int col_pattern_idx_end_of_aligned = (block->col_pattern_len / 4) * 4;
    for (int col_pattern_idx = 0; col_pattern_idx < col_pattern_idx_end_of_aligned; col_pattern_idx += 4) {

      float4 b0 = vector_load(&B_s[col_pattern_idx + 0][thd_x_vec_offset]);
      float4 b1 = vector_load(&B_s[col_pattern_idx + 1][thd_x_vec_offset]);
      float4 b2 = vector_load(&B_s[col_pattern_idx + 2][thd_x_vec_offset]);
      float4 b3 = vector_load(&B_s[col_pattern_idx + 3][thd_x_vec_offset]);

      int row_idx = (0 * blockDim.y + threadIdx.y);
      int row_offset = row_idx * block->col_pattern_len;
      float4 a = vector_load(&A_s[row_offset + col_pattern_idx + 0]);

      if (row_idx < block->num_rows) {
        FMAA(c0, a.x, b0);
        FMAA(c0, a.y, b1);
        FMAA(c0, a.z, b2);
        FMAA(c0, a.w, b3);
      }

      row_idx = (1 * blockDim.y + threadIdx.y);
      row_offset = row_idx * block->col_pattern_len;
      a = vector_load(&A_s[row_offset + col_pattern_idx + 0]);

      if (row_idx < block->num_rows) {
        FMAA(c1, a.x, b0);
        FMAA(c1, a.y, b1);
        FMAA(c1, a.z, b2);
        FMAA(c1, a.w, b3);
      }

      __syncthreads();
    }

    for (int col_pattern_idx = col_pattern_idx_end_of_aligned;
         col_pattern_idx < block->col_pattern_len; col_pattern_idx++) {

      int row_idx = (0 * blockDim.y + threadIdx.y);
      int row_offset = row_idx * block->col_pattern_len;
      if (row_idx < block->num_rows) {
        FMAA(c0, A_s[row_offset + col_pattern_idx + 0], &B_s[col_pattern_idx + 0][thd_x_vec_offset]);
      }

      row_idx = (1 * blockDim.y + threadIdx.y);
      row_offset = row_idx * block->col_pattern_len;
      if (row_idx < block->num_rows) {
        FMAA(c1, A_s[row_offset + col_pattern_idx + 0], &B_s[col_pattern_idx + 0][thd_x_vec_offset]);
      }

      __syncthreads();
    }

    int row_idx = (0 * blockDim.y + threadIdx.y);
    if (row_idx < block->num_rows) {
      vec_atomic_add_coeff(C, block->rows[row_idx], k + thd_x_vec_offset, c0);
    }

    row_idx = (1 * blockDim.y + threadIdx.y);
    if (row_idx < block->num_rows) {
      vec_atomic_add_coeff(C, block->rows[row_idx], k + thd_x_vec_offset, c1);
    }
  }
}

int CodeletMultiply::codelet_multiply(hipStream_t& stream, hipEvent_t& start, hipEvent_t& stop, const Block * blocks, size_t num_blocks,
                     const CSR<float>& A_h, const CSR<float>& A, const Dense& B, Dense& C) {
  dim3 grid_dim(num_blocks, 1);

  hipEventRecord(start, stream);
  _block_multiply<<<grid_dim, codelet_block_dim, 0, stream>>>(blocks, num_blocks, A, B, C);
  CHECK_CUDA(hipGetLastError());
  hipEventRecord(stop, stream);

  return 0;
}

