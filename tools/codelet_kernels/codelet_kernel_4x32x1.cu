#include "hip/hip_runtime.h"
//
// Created by lwilkinson on 12/7/21.
//

#include "codelet_kernel_utils.cuh"

using namespace CodeletMultiply;
using namespace codelet_4x32x1;

/// This example streams elementsPerThreadBlock worth of data from global memory
/// into a limited sized shared memory (elementsInShared) block to operate on in
/// multiple (two) stages. As stage N is kicked off, we can wait on and operate on stage N-1.
#include <hip/hip_runtime.h>
#include <cuda/barrier>
#include <hip/hip_cooperative_groups.h>
#include <cooperative_groups/memcpy_async.h>

namespace cg = cooperative_groups;

static dim3 codelet_block_dim(BLOCK_X_DIM, BLOCK_Y_DIM);

#define _load_a_reg_branchless(offset, reg) {                                                                   \
    a##reg = global_const_vector_load(&row_segment_values_ptr[threadIdx.y * MAX_ROWS_PER_BLOCK + offset]);      \
}

#define _load_a_reg_branchless(offset, reg) {                                                                   \
    a##reg = global_const_vector_load(&row_segment_values_ptr[threadIdx.y * MAX_ROWS_PER_BLOCK + offset]);      \
}

__global__ static void
_block_multiply_reg_storage_bypass_8x32x1(const Block * blocks, int num_blocks, const CSR<float> A, const Dense B, Dense C) {
  int block_idx = blockIdx.x;
  cg::thread_block tb = cg::this_thread_block();

  const Block *block = &blocks[block_idx];

  //cache(A);
  cache(B);
  cache(C);

  // TODO: make CODELET_MULTIPLY_MAX_COLS_PER_BLOCK dynamic
  //__shared__ __align__(32) float A_s[BLOCK_ROWS * BLOCK_COLS];
  __shared__ __align__(32) int A_rows[MAX_ROWS_PER_BLOCK];
  __shared__ __align__(32) int block_col_pattern[MAX_COLS_PER_BLOCK];
  __shared__ __align__(32) float B_s[MAX_COLS_PER_BLOCK][TILE_K];
  __shared__ __align__(32) float C_s[MAX_ROWS_PER_BLOCK][TILE_K];

  int block_col_pattern_len = block->col_pattern_len;
  int block_num_rows = block->num_rows;
  int* block_col_pattern_ptr = block->col_pattern;
  int * block_rows_ptr = block->rows;
  float* row_segment_values_ptr = block->row_segment_values;

  int non_zeros = block_num_rows * block_col_pattern_len;
  int thd_idx_linear = threadIdx.x + threadIdx.y * blockDim.x;
  int thd_x_vec_offset = threadIdx.x * VECTOR_WIDTH;
  int thd_linear_vec_offset = thd_idx_linear * VECTOR_WIDTH;
  int block_size = blockDim.x * blockDim.y;

  int vector_load_block_width = block_size * VECTOR_WIDTH;
  int a_vector_loads_full_block = non_zeros / vector_load_block_width;
  int a_vector_loads_partial_block_start = a_vector_loads_full_block * vector_load_block_width;
  int a_vector_partial_load = non_zeros - a_vector_loads_partial_block_start;


  int k = 0;

  cg::memcpy_async(tb, block_col_pattern,
                   cuda::aligned_size_t<16>(block_col_pattern_len),
                   block_col_pattern_ptr,
                   cuda::aligned_size_t<16>(block_col_pattern_len));


  cg::memcpy_async(tb, A_rows,
                   cuda::aligned_size_t<16>(block_num_rows),
                   block_rows_ptr,
                   cuda::aligned_size_t<16>(block_num_rows));


  cg::wait_prior<1>(tb); // Wait for block_col_pattern

#pragma unroll
  for (int i = 0; i < MAX_COLS_PER_BLOCK; i++) {
    cg::memcpy_async(tb, B_s[i],
                     cuda::aligned_size_t<16>(TILE_K),
                     coeff_ptr(B, block_col_pattern_ptr[i], k),
                     cuda::aligned_size_t<16>(B_cols - k));
  }

  __syncthreads();

  float4 a0, a1, a2, a3, a4, a5, a6, a7;
  float4 a8, a9, a10, a11, a12, a13, a14, a15;

  _load_a_reg_branchless( 0, 0);
  _load_a_reg_branchless( 4, 1);
  _load_a_reg_branchless( 8, 2);
  _load_a_reg_branchless(12, 3);
  _load_a_reg_branchless(16, 4);
  _load_a_reg_branchless(20, 5);
  _load_a_reg_branchless(24, 6);
  _load_a_reg_branchless(28, 7);

  _load_a_reg_branchless(MAX_ROWS_PER_BLOCK * 4 +  0, 8);
  _load_a_reg_branchless(MAX_ROWS_PER_BLOCK * 4 +  4, 9);
  _load_a_reg_branchless(MAX_ROWS_PER_BLOCK * 4 +  8, 10);
  _load_a_reg_branchless(MAX_ROWS_PER_BLOCK * 4 + 12, 11);
  _load_a_reg_branchless(MAX_ROWS_PER_BLOCK * 4 + 16, 12);
  _load_a_reg_branchless(MAX_ROWS_PER_BLOCK * 4 + 20, 13);
  _load_a_reg_branchless(MAX_ROWS_PER_BLOCK * 4 + 24, 14);
  _load_a_reg_branchless(MAX_ROWS_PER_BLOCK * 4 + 28, 15);

  cg::wait_prior<1>(tb); // Wait for A_rows

  for (int k = 0; k < B_cols; k+= TILE_K) {
#pragma unroll
    for (int i = 0; i < MAX_COLS_PER_BLOCK; i++) {
      cg::memcpy_async(tb, B_s[i], cuda::aligned_size_t<16>(TILE_K),
                       coeff_ptr(B, block_col_pattern_ptr[i], k),
                       cuda::aligned_size_t<16>(B_cols - k));
    }

    cg::wait_prior<MAX_COLS_PER_BLOCK>(tb); //  Wait for previous B_s batch

    int offset = 0;

    float4 c0 = { .x = 0, .y = 0, .z = 0, .w = 0 };
    float4 c1 = { .x = 0, .y = 0, .z = 0, .w = 0 };

    float4 b0 = shared_const_vector_load(&B_s[offset + 0][threadIdx.x * VECTOR_WIDTH]);
    float4 b1 = shared_const_vector_load(&B_s[offset + 1][threadIdx.x * VECTOR_WIDTH]);
    float4 b2 = shared_const_vector_load(&B_s[offset + 2][threadIdx.x * VECTOR_WIDTH]);
    float4 b3 = shared_const_vector_load(&B_s[offset + 3][threadIdx.x * VECTOR_WIDTH]);
    float4 b4 = shared_const_vector_load(&B_s[offset + 4][threadIdx.x * VECTOR_WIDTH]);
    float4 b5 = shared_const_vector_load(&B_s[offset + 5][threadIdx.x * VECTOR_WIDTH]);
    float4 b6 = shared_const_vector_load(&B_s[offset + 6][threadIdx.x * VECTOR_WIDTH]);
    float4 b7 = shared_const_vector_load(&B_s[offset + 7][threadIdx.x * VECTOR_WIDTH]);

    FMAA(c0, a0.x, b0);
    FMAA(c0, a0.y, b1);
    FMAA(c0, a0.z, b2);
    FMAA(c0, a0.w, b3);
    FMAA(c0, a1.x, b4);
    FMAA(c0, a1.y, b5);
    FMAA(c0, a1.z, b6);
    FMAA(c0, a1.w, b7);

    FMAA(c1, a8.x, b0);
    FMAA(c1, a8.y, b1);
    FMAA(c1, a8.z, b2);
    FMAA(c1, a8.w, b3);
    FMAA(c1, a9.x, b4);
    FMAA(c1, a9.y, b5);
    FMAA(c1, a9.z, b6);
    FMAA(c1, a9.w, b7);

    offset = 8;

    b0 = shared_const_vector_load(&B_s[offset + 0][threadIdx.x * VECTOR_WIDTH]);
    b1 = shared_const_vector_load(&B_s[offset + 1][threadIdx.x * VECTOR_WIDTH]);
    b2 = shared_const_vector_load(&B_s[offset + 2][threadIdx.x * VECTOR_WIDTH]);
    b3 = shared_const_vector_load(&B_s[offset + 3][threadIdx.x * VECTOR_WIDTH]);
    b4 = shared_const_vector_load(&B_s[offset + 4][threadIdx.x * VECTOR_WIDTH]);
    b5 = shared_const_vector_load(&B_s[offset + 5][threadIdx.x * VECTOR_WIDTH]);
    b6 = shared_const_vector_load(&B_s[offset + 6][threadIdx.x * VECTOR_WIDTH]);
    b7 = shared_const_vector_load(&B_s[offset + 7][threadIdx.x * VECTOR_WIDTH]);

    FMAA(c0, a2.x, b0);
    FMAA(c0, a2.y, b1);
    FMAA(c0, a2.z, b2);
    FMAA(c0, a2.w, b3);
    FMAA(c0, a3.x, b4);
    FMAA(c0, a3.y, b5);
    FMAA(c0, a3.z, b6);
    FMAA(c0, a3.w, b7);

    FMAA(c1, a10.x, b0);
    FMAA(c1, a10.y, b1);
    FMAA(c1, a10.z, b2);
    FMAA(c1, a10.w, b3);
    FMAA(c1, a11.x, b4);
    FMAA(c1, a11.y, b5);
    FMAA(c1, a11.z, b6);
    FMAA(c1, a11.w, b7);

    offset = 16;

    b0 = shared_const_vector_load(&B_s[offset + 0][threadIdx.x * VECTOR_WIDTH]);
    b1 = shared_const_vector_load(&B_s[offset + 1][threadIdx.x * VECTOR_WIDTH]);
    b2 = shared_const_vector_load(&B_s[offset + 2][threadIdx.x * VECTOR_WIDTH]);
    b3 = shared_const_vector_load(&B_s[offset + 3][threadIdx.x * VECTOR_WIDTH]);
    b4 = shared_const_vector_load(&B_s[offset + 4][threadIdx.x * VECTOR_WIDTH]);
    b5 = shared_const_vector_load(&B_s[offset + 5][threadIdx.x * VECTOR_WIDTH]);
    b6 = shared_const_vector_load(&B_s[offset + 6][threadIdx.x * VECTOR_WIDTH]);
    b7 = shared_const_vector_load(&B_s[offset + 7][threadIdx.x * VECTOR_WIDTH]);

    FMAA(c0, a4.x, b0);
    FMAA(c0, a4.y, b1);
    FMAA(c0, a4.z, b2);
    FMAA(c0, a4.w, b3);
    FMAA(c0, a5.x, b4);
    FMAA(c0, a5.y, b5);
    FMAA(c0, a5.z, b6);
    FMAA(c0, a5.w, b7);

    FMAA(c1, a12.x, b0);
    FMAA(c1, a12.y, b1);
    FMAA(c1, a12.z, b2);
    FMAA(c1, a12.w, b3);
    FMAA(c1, a13.x, b4);
    FMAA(c1, a13.y, b5);
    FMAA(c1, a13.z, b6);
    FMAA(c1, a13.w, b7);

    offset = 24;

    b0 = shared_const_vector_load(&B_s[offset + 0][threadIdx.x * VECTOR_WIDTH]);
    b1 = shared_const_vector_load(&B_s[offset + 1][threadIdx.x * VECTOR_WIDTH]);
    b2 = shared_const_vector_load(&B_s[offset + 2][threadIdx.x * VECTOR_WIDTH]);
    b3 = shared_const_vector_load(&B_s[offset + 3][threadIdx.x * VECTOR_WIDTH]);
    b4 = shared_const_vector_load(&B_s[offset + 4][threadIdx.x * VECTOR_WIDTH]);
    b5 = shared_const_vector_load(&B_s[offset + 5][threadIdx.x * VECTOR_WIDTH]);
    b6 = shared_const_vector_load(&B_s[offset + 6][threadIdx.x * VECTOR_WIDTH]);
    b7 = shared_const_vector_load(&B_s[offset + 7][threadIdx.x * VECTOR_WIDTH]);

    FMAA(c0, a6.x, b0);
    FMAA(c0, a6.y, b1);
    FMAA(c0, a6.z, b2);
    FMAA(c0, a6.w, b3);
    FMAA(c0, a7.x, b4);
    FMAA(c0, a7.y, b5);
    FMAA(c0, a7.z, b6);
    FMAA(c0, a7.w, b7);

    FMAA(c1, a14.x, b0);
    FMAA(c1, a14.y, b1);
    FMAA(c1, a14.z, b2);
    FMAA(c1, a14.w, b3);
    FMAA(c1, a15.x, b4);
    FMAA(c1, a15.y, b5);
    FMAA(c1, a15.z, b6);
    FMAA(c1, a15.w, b7);

    shared_vector_store(&C_s[threadIdx.y][thd_x_vec_offset], c0);
    shared_vector_store(&C_s[4 + threadIdx.y][thd_x_vec_offset], c1);

    for (int row_idx = 0; row_idx < MAX_ROWS_PER_BLOCK; row_idx++) {
      float *C_addr_g = coeff_ptr(C, A_rows[row_idx], k + thd_idx_linear);
      float C_to_add = C_s[row_idx][thd_idx_linear];
      atomicAdd(C_addr_g, C_to_add);
    }
  }
}


#define _load_b_bypass(base_idx, idx)                                                \
    cg::memcpy_async(tb, B_async[idx],                                               \
            cuda::aligned_size_t<16>(TILE_K),                                        \
            coeff_ptr(B, block_col_pattern[col_pattern_idx + base_idx + idx], k),    \
            cuda::aligned_size_t<16>(TILE_K));

#define _load_b_reg_from_bypass(reg, idx)                                           \
    b##reg = shared_const_vector_load(&B_async[idx][thd_x_vec_offset]);             \


__global__ void _block_multiply_reg_storage_orig_4x32x1(const Block * blocks,
                                                        int num_blocks,
                                                        const CSR<float> A,
                                                        const Dense B, Dense C) {
  int block_idx = blockIdx.x;
  cg::thread_block tb = cg::this_thread_block();

  const Block *block = &blocks[block_idx];

  //cache(A);
  cache(B);
  cache(C);

  // TODO: make CODELET_MULTIPLY_MAX_COLS_PER_BLOCK dynamic
  __shared__ __align__(32) float A_s[MAX_ROWS_PER_BLOCK * MAX_COLS_PER_BLOCK];
  __shared__ __align__(32) int A_rows[MAX_ROWS_PER_BLOCK];
  __shared__ __align__(32) int block_col_pattern[MAX_COLS_PER_BLOCK];
  //__shared__ __align__(32) float B_s[BLOCK_COLS][TILE_K];

  __shared__ __align__(32) float B_async[MAX_COLS_PER_BLOCK / 2][TILE_K];
  __shared__ __align__(32) float C_s[MAX_ROWS_PER_BLOCK][TILE_K];

  int block_col_pattern_len = block->col_pattern_len;
  int block_num_rows = block->num_rows;
  int* block_col_pattern_ptr = block->col_pattern;
  int * block_rows_ptr = block->rows;
  float* row_segment_values_ptr = block->row_segment_values;

  int non_zeros = block_num_rows * block_col_pattern_len;
  int thd_idx_linear = threadIdx.x + threadIdx.y * blockDim.x;
  int thd_x_vec_offset = threadIdx.x * VECTOR_WIDTH;
  int thd_linear_vec_offset = thd_idx_linear * VECTOR_WIDTH;
  int block_size = blockDim.x * blockDim.y;

  int vector_load_block_width = block_size * VECTOR_WIDTH;
  int a_vector_loads_full_block = non_zeros / vector_load_block_width;
  int a_vector_loads_partial_block_start = a_vector_loads_full_block * vector_load_block_width;
  int a_vector_partial_load = non_zeros - a_vector_loads_partial_block_start;

  cg::memcpy_async(tb, block_col_pattern,
                   cuda::aligned_size_t<16>(block_col_pattern_len),
                   block_col_pattern_ptr,
                   cuda::aligned_size_t<16>(block_col_pattern_len));

  cg::memcpy_async(tb, A_s,
                   cuda::aligned_size_t<16>(non_zeros),
                   row_segment_values_ptr,
                   cuda::aligned_size_t<16>(non_zeros));

  cg::memcpy_async(tb, A_rows,
                   cuda::aligned_size_t<16>(block_num_rows),
                   block_rows_ptr,
                   cuda::aligned_size_t<16>(block_num_rows));

  cg::wait_prior<1>(tb); // Wait for block_col_pattern

  float4 a0;
  float4 a1;
  float4 a2;
  float4 a3;
  float4 a4;
  float4 a5;
  float4 a6;
  float4 a7;

  int row_idx0 = (0 * blockDim.y + threadIdx.y);
  int row_offset0 = row_idx0 * block_col_pattern_len;

  for (int k = 0; k < B_cols; k+= TILE_K) {

    static_assert(MAX_ROWS_PER_BLOCK / BLOCK_Y_DIM  == 1);
    float4 c0 = { .x = 0, .y = 0, .z = 0, .w = 0 };

    float4 b0, b1, b2, b3, b4, b5, b6, b7;
    float4 b8, b9, b10, b11, b12, b13, b14, b15;

    float4 b16, b17, b18, b19, b20, b21, b22, b23;
    float4 b24, b25, b26, b27, b28, b29, b30, b31;

    _load_b_reg_branchless(0, 0, 0);
    _load_b_reg_branchless(0, 1, 1);
    _load_b_reg_branchless(0, 2, 2);
    _load_b_reg_branchless(0, 3, 3);
    _load_b_reg_branchless(0, 4, 4);
    _load_b_reg_branchless(0, 5, 5);
    _load_b_reg_branchless(0, 6, 6);
    _load_b_reg_branchless(0, 7, 7);

    _load_b_reg_branchless(0, 8, 8);
    _load_b_reg_branchless(0, 9, 9);
    _load_b_reg_branchless(0, 10, 10);
    _load_b_reg_branchless(0, 11, 11);
    _load_b_reg_branchless(0, 12, 12);
    _load_b_reg_branchless(0, 13, 13);
    _load_b_reg_branchless(0, 14, 14);
    _load_b_reg_branchless(0, 15, 15);

    if (k == 0) {
      cg::wait_prior<2>(tb); // Wait for A_s, A_rows

      a0 = shared_const_vector_load(&A_s[row_offset0 + 0]);
      a1 = shared_const_vector_load(&A_s[row_offset0 + 4]);
      a2 = shared_const_vector_load(&A_s[row_offset0 + 8]);
      a3 = shared_const_vector_load(&A_s[row_offset0 + 12]);
      a4 = shared_const_vector_load(&A_s[row_offset0 + 16]);
      a5 = shared_const_vector_load(&A_s[row_offset0 + 20]);
      a6 = shared_const_vector_load(&A_s[row_offset0 + 24]);
      a7 = shared_const_vector_load(&A_s[row_offset0 + 28]);
    }

    _load_b_reg_branchless(0, 16, 16);
    _load_b_reg_branchless(0, 17, 17);
    _load_b_reg_branchless(0, 18, 18);
    _load_b_reg_branchless(0, 19, 19);
    _load_b_reg_branchless(0, 20, 20);
    _load_b_reg_branchless(0, 21, 21);
    _load_b_reg_branchless(0, 22, 22);
    _load_b_reg_branchless(0, 23, 23);

    _load_b_reg_branchless(0, 24, 24);
    _load_b_reg_branchless(0, 25, 25);
    _load_b_reg_branchless(0, 26, 26);
    _load_b_reg_branchless(0, 27, 27);
    _load_b_reg_branchless(0, 28, 28);
    _load_b_reg_branchless(0, 29, 29);
    _load_b_reg_branchless(0, 30, 30);
    _load_b_reg_branchless(0, 31, 31);

    FMAA(c0, a0.x, b0);
    FMAA(c0, a0.y, b1);
    FMAA(c0, a0.z, b2);
    FMAA(c0, a0.w, b3);
    FMAA(c0, a1.x, b4);
    FMAA(c0, a1.y, b5);
    FMAA(c0, a1.z, b6);
    FMAA(c0, a1.w, b7);

    FMAA(c0, a2.x, b8);
    FMAA(c0, a2.y, b9);
    FMAA(c0, a2.z, b10);
    FMAA(c0, a2.w, b11);
    FMAA(c0, a3.x, b12);
    FMAA(c0, a3.y, b13);
    FMAA(c0, a3.z, b14);
    FMAA(c0, a3.w, b15);

    FMAA(c0, a4.x, b16);
    FMAA(c0, a4.y, b17);
    FMAA(c0, a4.z, b18);
    FMAA(c0, a4.w, b19);
    FMAA(c0, a5.x, b20);
    FMAA(c0, a5.y, b21);
    FMAA(c0, a5.z, b22);
    FMAA(c0, a5.w, b23);

    FMAA(c0, a6.x, b24);
    FMAA(c0, a6.y, b25);
    FMAA(c0, a6.z, b26);
    FMAA(c0, a6.w, b27);
    FMAA(c0, a7.x, b28);
    FMAA(c0, a7.y, b29);
    FMAA(c0, a7.z, b30);
    FMAA(c0, a7.w, b31);

    shared_vector_store(&C_s[row_idx0][thd_x_vec_offset], c0);

    __syncthreads();

    for (int row_idx = 0; row_idx < MAX_ROWS_PER_BLOCK; row_idx++) {
      float value = C_s[row_idx][thd_idx_linear];
      float* C_add_addr = coeff_ptr(C, A_rows[row_idx], k + thd_idx_linear);
      atomicAdd(C_add_addr, value);
    }
  }
}

int
codelet_4x32x1::codelet_multiply(hipStream_t &stream, hipEvent_t &start, hipEvent_t &stop, const Block *blocks,
                                size_t num_blocks,
                                const CSR<float> &A_h, const CSR<float> &A, const Dense &B, Dense &C) {
  dim3 grid_dim(num_blocks, 1);

  hipFuncSetCacheConfig(reinterpret_cast<const void*>(_block_multiply_reg_storage_orig_4x32x1), hipFuncCachePreferShared);
  hipEventRecord(start, stream);
  _block_multiply_reg_storage_orig_4x32x1<<<grid_dim, codelet_block_dim, 0, stream>>>(blocks, num_blocks, A, B, C);
  CHECK_CUDA(hipGetLastError());
  hipEventRecord(stop, stream);

  return 0;
}