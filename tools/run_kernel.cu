//
// Created by lwilkinson on 12/7/21.
//

#include "run_kernel.cuh"


int run_kernel(const Dense& A, const Dense& B, const Dense& C, const std::string& name, test_harness::csv_row_t &csv_row,
               int(*kernel)(hipStream_t& stream, hipEvent_t& start, hipEvent_t& stop, const Dense& A, const Dense& B, Dense& C)) {
  float *A_values_d, *B_values_d, *C_values_d;

  CHECK_CUDA(hipMalloc(&A_values_d, A.rows * A.cols * sizeof(float)));
  CHECK_CUDA(hipMalloc(&B_values_d, B.rows * B.cols * sizeof(float)));
  CHECK_CUDA(hipMalloc(&C_values_d, C.rows * C.cols * sizeof(float)));

  CHECK_CUDA(hipMemcpy(A_values_d, A.values, A.rows * A.cols * sizeof(float), hipMemcpyHostToDevice));
  CHECK_CUDA(hipMemcpy(B_values_d, B.values, B.rows * B.cols * sizeof(float), hipMemcpyHostToDevice));

  Dense A_d = A;
  Dense B_d = B;
  Dense C_d = C;

  A_d.values = A_values_d;
  B_d.values = B_values_d;
  C_d.values = C_values_d;

  hipEvent_t start, stop;
  hipEventCreate(&start);
  hipEventCreate(&stop);

  hipStream_t stream = NULL;
  CHECK_CUDA(hipStreamCreate(&stream))

  hipDeviceSynchronize();

  float total_time = 0;
  for (int iter = 0; iter < ITERATIONS + 1; iter++) {
    CHECK_CUDA(hipMemset(C_values_d, 0, C.rows * C.cols * sizeof(float)));
    hipDeviceSynchronize();

    kernel(stream, start, stop, A_d, B_d, C_d);
    hipEventSynchronize(stop);
    CHECK_CUDA(hipGetLastError());

    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);

    if (iter >= 1) total_time += milliseconds; // Skip a warm up
  }

  std::cout << name << " took " << total_time / ITERATIONS << "ms (avg)" << std::endl;
  test_harness::csv_row_insert(csv_row, name, total_time / ITERATIONS);

  hipDeviceSynchronize();

  CHECK_CUDA(hipMemcpy(C.values, C_values_d, C.rows * C.cols * sizeof(float), hipMemcpyDeviceToHost));

  CHECK_CUDA(hipFree(A_values_d))
  CHECK_CUDA(hipFree(B_values_d))
  CHECK_CUDA(hipFree(C_values_d))

  hipDeviceSynchronize();

  hipEventDestroy(start);
  hipEventDestroy(stop);
  hipStreamDestroy(stream);

  hipDeviceSynchronize();

  return 0;
}

int run_kernel(const CSR<float>& A, const Dense& B, Dense& C, const std::string& name, test_harness::csv_row_t &csv_row,
               int(*kernel)(hipStream_t& stream, hipEvent_t& start, hipEvent_t& stop, const CSR<float>& A_h, const CSR<float>& A, const Dense& B, Dense& C)) {
  float *A_values_d, *B_values_d, *C_values_d;
  int *A_row_offsets_d, *A_col_indices_d;

  CHECK_CUDA(hipMalloc(&A_values_d, A.nnz * sizeof(float)));
  CHECK_CUDA(hipMalloc(&A_row_offsets_d, (A.rows + 1) * sizeof(int)));
  CHECK_CUDA(hipMalloc(&A_col_indices_d, A.nnz * sizeof(int)));

  CHECK_CUDA(hipMalloc(&B_values_d, B.rows * B.cols * sizeof(float)));
  CHECK_CUDA(hipMalloc(&C_values_d, C.rows * C.cols * sizeof(float)));

  CHECK_CUDA(hipMemcpy(A_values_d, A.values, A.nnz * sizeof(float), hipMemcpyHostToDevice));
  CHECK_CUDA(hipMemcpy(A_row_offsets_d, A.row_offsets, (A.rows + 1) * sizeof(int), hipMemcpyHostToDevice));
  CHECK_CUDA(hipMemcpy(A_col_indices_d, A.col_indices, A.nnz * sizeof(int), hipMemcpyHostToDevice));

  CHECK_CUDA(hipMemcpy(B_values_d, B.values, B.rows * B.cols * sizeof(float), hipMemcpyHostToDevice));

  CSR<float> A_d = A;
  Dense B_d = B;
  Dense C_d = C;

  A_d.values = A_values_d;
  A_d.row_offsets = A_row_offsets_d;
  A_d.col_indices = A_col_indices_d;
  B_d.values = B_values_d;
  C_d.values = C_values_d;

  hipDeviceSynchronize();

  hipEvent_t start, stop;
  hipEventCreate(&start);
  hipEventCreate(&stop);

  hipStream_t stream = NULL;
  CHECK_CUDA(hipStreamCreate(&stream))

  hipDeviceSynchronize();

  float total_time = 0;
  for (int iter = 0; iter < ITERATIONS + 1; iter++) {
    CHECK_CUDA(hipMemset(C_values_d, 0, C.rows * C.cols * sizeof(float)));
    hipDeviceSynchronize();

    kernel(stream, start, stop, A, A_d, B_d, C_d);
    hipEventSynchronize(stop);
    CHECK_CUDA(hipGetLastError());

    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);

    if (iter >= 1) total_time += milliseconds; // Skip a warm up
  }

  std::cout << name << " took " << total_time / ITERATIONS << "ms (avg)" << std::endl;
  test_harness::csv_row_insert(csv_row, name, total_time / ITERATIONS);

  CHECK_CUDA(hipMemcpy(C.values, C_values_d, C.rows * C.cols * sizeof(float), hipMemcpyDeviceToHost));

  hipDeviceSynchronize();

  CHECK_CUDA(hipFree(A_values_d));
  CHECK_CUDA(hipFree(A_row_offsets_d));
  CHECK_CUDA(hipFree(A_col_indices_d));

  CHECK_CUDA(hipFree(B_values_d))
  CHECK_CUDA(hipFree(C_values_d))

  hipDeviceSynchronize();

  hipEventDestroy(start);
  hipEventDestroy(stop);
  hipStreamDestroy(stream);

  hipDeviceSynchronize();

  return 0;
}

int run_kernel(const std::vector<CodeletMultiply::Block> &blocks, const CSR<float>& A, const Dense& B, Dense& C, const std::string& name, test_harness::csv_row_t &csv_row,
               int(*kernel)(hipStream_t& stream, hipEvent_t& start, hipEvent_t& stop, const CodeletMultiply::Block * blocks, size_t num_blocks, const CSR<float>& A_h, const CSR<float>& A, const Dense& B, Dense& C)) {
  float *A_values_d, *B_values_d, *C_values_d;
  int *A_row_offsets_d, *A_col_indices_d;
  std::vector<CodeletMultiply::Block> blocks_d_temp(blocks);
  CodeletMultiply::Block* blocks_d;

  CHECK_CUDA(hipMalloc(&A_values_d, A.nnz * sizeof(float)));
  CHECK_CUDA(hipMalloc(&A_row_offsets_d, (A.rows + 1) * sizeof(int)));
  CHECK_CUDA(hipMalloc(&A_col_indices_d, A.nnz * sizeof(int)));

  for (int i = 0; i < blocks.size(); i++) {
    size_t num_rows = blocks_d_temp[i].num_rows * sizeof(int) * blocks[0].batch_size;
    CHECK_CUDA(hipMalloc(&blocks_d_temp[i].rows, num_rows));
    CHECK_CUDA(hipMemcpy(blocks_d_temp[i].rows, blocks[i].rows, num_rows, hipMemcpyHostToDevice));

    size_t col_pattern_len = blocks_d_temp[i].col_pattern_len * sizeof(int) * blocks[0].batch_size;
    CHECK_CUDA(hipMalloc(&blocks_d_temp[i].col_pattern, col_pattern_len));
    CHECK_CUDA(hipMemcpy(blocks_d_temp[i].col_pattern, blocks[i].col_pattern, col_pattern_len, hipMemcpyHostToDevice));

    size_t values_len = blocks_d_temp[i].num_rows * blocks_d_temp[i].col_pattern_len * sizeof(float) * blocks[0].batch_size;
    CHECK_CUDA(hipMalloc(&blocks_d_temp[i].row_segment_values, values_len));
    CHECK_CUDA(hipMemcpy(blocks_d_temp[i].row_segment_values, blocks[i].row_segment_values, values_len, hipMemcpyHostToDevice));
  }

  CHECK_CUDA(hipMalloc(&blocks_d, blocks_d_temp.size() * sizeof(CodeletMultiply::Block)));
  CHECK_CUDA(hipMemcpy(blocks_d, blocks_d_temp.data(), blocks_d_temp.size() * sizeof(CodeletMultiply::Block), hipMemcpyHostToDevice));

  CHECK_CUDA(hipMalloc(&B_values_d, B.rows * B.cols * sizeof(float)));
  CHECK_CUDA(hipMalloc(&C_values_d, C.rows * C.cols * sizeof(float)));

  CHECK_CUDA(hipMemcpy(A_values_d, A.values, A.nnz * sizeof(float), hipMemcpyHostToDevice));
  CHECK_CUDA(hipMemcpy(A_row_offsets_d, A.row_offsets, (A.rows + 1) * sizeof(int), hipMemcpyHostToDevice));
  CHECK_CUDA(hipMemcpy(A_col_indices_d, A.col_indices, A.nnz * sizeof(int), hipMemcpyHostToDevice));

  CHECK_CUDA(hipMemcpy(B_values_d, B.values, B.rows * B.cols * sizeof(float), hipMemcpyHostToDevice));
  CHECK_CUDA(hipMemset(C_values_d, 0, C.rows * C.cols * sizeof(float)));

  CSR<float> A_d = A;
  Dense B_d = B;
  Dense C_d = C;

  A_d.values = A_values_d;
  A_d.row_offsets = A_row_offsets_d;
  A_d.col_indices = A_col_indices_d;
  B_d.values = B_values_d;
  C_d.values = C_values_d;

  hipDeviceSynchronize();

  hipEvent_t start, stop;
  hipEventCreate(&start);
  hipEventCreate(&stop);

  hipStream_t stream = NULL;
  CHECK_CUDA(hipStreamCreate(&stream))

  hipDeviceSynchronize();

  float total_time = 0;
  for (int iter = 0; iter < ITERATIONS + 1; iter++) {
    CHECK_CUDA(hipMemset(C_values_d, 0, C.rows * C.cols * sizeof(float)));
    hipDeviceSynchronize();

    kernel(stream, start, stop, blocks_d, blocks.size(), A, A_d, B_d, C_d);
    CHECK_CUDA(hipGetLastError());
    hipDeviceSynchronize();
    CHECK_CUDA(hipGetLastError());
    hipEventSynchronize(stop);
    CHECK_CUDA(hipGetLastError());

    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);

    if (iter >= 1) total_time += milliseconds; // Skip a warm up
  }

  std::cout << name << " took " << total_time / ITERATIONS << "ms (avg)" << std::endl;
  test_harness::csv_row_insert(csv_row, name, total_time / ITERATIONS);

  CHECK_CUDA(hipMemcpy(C.values, C_values_d, C.rows * C.cols * sizeof(float), hipMemcpyDeviceToHost));

  hipDeviceSynchronize();

  CHECK_CUDA(hipFree(A_values_d));
  CHECK_CUDA(hipFree(A_row_offsets_d));
  CHECK_CUDA(hipFree(A_col_indices_d));

  for (int i = 0; i < blocks.size(); i++) {
    CHECK_CUDA(hipFree(blocks_d_temp[i].rows));
    CHECK_CUDA(hipFree(blocks_d_temp[i].col_pattern));
    CHECK_CUDA(hipFree(blocks_d_temp[i].row_segment_values));
  }

  CHECK_CUDA(hipFree(B_values_d));
  CHECK_CUDA(hipFree(C_values_d));

  hipDeviceSynchronize();

  hipEventDestroy(start);
  hipEventDestroy(stop);
  hipStreamDestroy(stream);

  hipDeviceSynchronize();

  return 0;
}
